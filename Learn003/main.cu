
#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>

const static int blockSize = 256;

__global__ void AddVec(const float* A,const float* B, float* C ,int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<size)
    {
        C[i] = A[i] - B[i];
    }
    
}

int main(int argc,char** argv)
{
    hipError_t err = hipSuccess;
    int num  = 3000000;
    int size = num* sizeof(float);
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);

    for (int i = 0; i < num; i++)
    {
        h_A[i] = (double)rand() / (RAND_MAX +1.0);
        h_B[i] = (double)rand() / (RAND_MAX +1.0);
    }
    float* h_C = (float*)malloc(size);
    for (int i = 0; i < num; i++)
    {
        h_C[i] = h_A[i] - h_B[i];
        // printf("the h_A[%d]+h_B[%d] = h_C[%d] { %f+%f=%f} \n",i,i,i,h_A[i],h_B[i],h_C[i]);
    }
    
    
    float* d_A;
    err = hipMalloc((void **)&d_A, size);
    float* d_B;
    err = hipMalloc((void **)&d_B,size);
    float* d_C;
    err = hipMalloc((void **)&d_C,size);

    err = hipMemcpy(d_A,h_A, size, hipMemcpyKind::hipMemcpyHostToDevice);
    err = hipMemcpy(d_B,h_B,size,hipMemcpyKind::hipMemcpyHostToDevice);

    int blocknum = blockSize;
    int gridnum = (num + blockSize - 1) / (blockSize);
    AddVec<<<gridnum, blocknum>>>(d_A, d_B, d_C, num);

    float* h_C1 = (float*)malloc(size);
    err = hipMemcpy(h_C1,d_C,size,hipMemcpyKind::hipMemcpyDeviceToHost);
    for (int i = 0; i < num; i++)
    {
        if (fabs(h_A[i] - h_B[i] - h_C1[i]) > 1e-5)
        {
            printf("error!!!\n");
            break;
        }
    }
    
    err = hipFree(d_A);
    err = hipFree(d_B);
    err = hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
    printf("第一个程序!!!");
}